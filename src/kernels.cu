#include "hip/hip_runtime.h"
#include "kernels.cuh"

////// DEVICE FUNCTIONS

__device__ inline unsigned short count_moore_neighbours(bool *grid,
                                                        unsigned int rows,
                                                        unsigned int cols,
                                                        unsigned int idx) {
    unsigned short livingNeighbours = 0;
    // we can calculate the neighbours directly since we're using safety borders
    unsigned int neighbours[] = {
        idx - cols - 1, // top left
        idx - cols,     // top center
        idx - cols + 1, // top right
        idx - 1,        // middle left
        idx + 1,        // middle right
        idx + cols - 1, // bottom left
        idx + cols,     // bottom center
        idx + cols + 1, // bottom right
    };
    for (unsigned short nidx = 0; nidx < 8; nidx++)
        if (grid[neighbours[nidx]])
            livingNeighbours++;

    return livingNeighbours;
}

__device__ inline bool game_of_life(bool isAlive,
                                    unsigned short livingNeighbours) {
    // 1. Any live cell with two or three live neighbours survives.
    if (isAlive)
        return livingNeighbours == 2 || livingNeighbours == 3;
    // 2. Any dead cell with three live neighbours becomes a live cell.
    else if (livingNeighbours == 3)
        return true;
    // 3. All other live cells die in the next generation. Similarly,
    // all other dead cells stay dead.
    else
        return false;
}

////// CUDA KERNELS

__global__ void k_setup_rng(unsigned int rows, unsigned int cols,
                            hiprandState *globalRandState, unsigned long seed) {
    dim3 stride(gridDim.x * blockDim.x, gridDim.y * blockDim.x);

    // note: we're using safety borders
    for (unsigned int y = blockDim.y * blockIdx.y + threadIdx.y + 1;
         y < rows - 1; y += stride.y) {
        for (unsigned int x = blockDim.x * blockIdx.x + threadIdx.x + 1;
             x < cols - 1; x += stride.x) {
            unsigned int idx = y * cols + x;
            hiprand_init(seed, idx, 0, &globalRandState[idx]);
        }
    }
}

__global__ void k_init_grid(bool *grid, unsigned int rows, unsigned int cols,
                            hiprandState *globalRandState,
                            float spawnProbability) {
    dim3 stride(gridDim.x * blockDim.x, gridDim.y * blockDim.x);

    // note: we're using safety borders
    for (unsigned int y = blockDim.y * blockIdx.y + threadIdx.y + 1;
         y < rows - 1; y += stride.y) {
        for (unsigned int x = blockDim.x * blockIdx.x + threadIdx.x + 1;
             x < cols - 1; x += stride.x) {
            unsigned int idx = y * cols + x;
            grid[idx] =
                hiprand_uniform(&globalRandState[idx]) < spawnProbability;
        }
    }
}

__global__ void k_update_grid_buffers(bool *grid, fvec2s *gridVertices,
                                      unsigned int rows, unsigned int cols,
                                      unsigned int numVerticesX,
                                      uvec2 cellDensity) {
    dim3 stride(gridDim.x * blockDim.x, gridDim.y * blockDim.x);

    // note: we're using safety borders
    for (unsigned int y = blockDim.y * blockIdx.y + threadIdx.y + 1;
         y < rows - 1; y += stride.y) {
        for (unsigned int x = blockDim.x * blockIdx.x + threadIdx.x + 1;
             x < cols - 1; x += stride.x) {
            // try avoiding further operations when not needed
            if (grid[y * cols + x]) {
                unsigned int vx = x / cellDensity.x;
                unsigned int vy = y / cellDensity.y;
                // no need to be atomic on this read
                if (gridVertices[vy * numVerticesX + vx].state == 0)
                    atomicMax(&gridVertices[vy * numVerticesX + vx].state,
                              (int)grid[y * cols + x]);
            }
        }
    }
}

__global__ void k_reset_grid_buffers(fvec2s *gridVertices,
                                     unsigned int numVerticesX,
                                     unsigned int numVerticesY) {
    dim3 stride(gridDim.x * blockDim.x, gridDim.y * blockDim.x);

    for (unsigned int y = blockDim.y * blockIdx.y + threadIdx.y;
         y < numVerticesY; y += stride.y) {
        for (unsigned int x = blockDim.x * blockIdx.x + threadIdx.x;
             x < numVerticesX; x += stride.x) {
            gridVertices[y * numVerticesX + x].state = 0;
        }
    }
}

__global__ void k_compute_grid_count_rule(bool *grid, bool *nextGrid,
                                          unsigned int rows, unsigned int cols,
                                          hiprandState *globalRandState,
                                          float virtualSpawnProbability,
                                          bool countAliveCells,
                                          unsigned int *activeCellCount) {
    dim3 stride(gridDim.x * blockDim.x, gridDim.y * blockDim.x);

    // note: we're using safety borders
    for (unsigned int y = blockDim.y * blockIdx.y + threadIdx.y + 1;
         y < rows - 1; y += stride.y) {
        for (unsigned int x = blockDim.x * blockIdx.x + threadIdx.x + 1;
             x < cols - 1; x += stride.x) {
            unsigned int idx = y * cols + x;
            bool newState = false;

            // 0. Virtual particle spawn probability
            if (virtualSpawnProbability > 0 &&
                hiprand_uniform(&globalRandState[idx]) < virtualSpawnProbability)
                newState = true;
            else
                newState = game_of_life(
                    grid[idx], count_moore_neighbours(grid, rows, cols, idx));

            // avoid atomicAdd when not necessary
            if (countAliveCells && newState)
                atomicAdd(activeCellCount, 1);

            nextGrid[idx] = newState;
        }
    }
}
