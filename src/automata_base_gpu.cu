#include "hip/hip_runtime.h"
// isolate cuda specific imports
#include <chrono>
#include <cuda_gl_interop.h>
#include <sstream>
#include <spdlog/spdlog.h>
#include <iostream>

#include "automata_base_gpu.cuh"
#include "stats.hpp"

namespace gpu {

AutomataBase::AutomataBase(const unsigned long randSeed,
                           std::ostringstream *const pLiveLogBuffer,
                           const uint *const gridVBO) {
    spdlog::info("Initializing automata GPU engine...");

    hipDeviceProp_t gpuProps;
    hipGetDevice(&mGpuDeviceId);
    hipGetDeviceProperties(&gpuProps, mGpuDeviceId);
    CUDA_ASSERT(hipDeviceSetCacheConfig(hipFuncCachePreferL1))

    // allocate memory
    // gpu only, not managed
    CUDA_ASSERT(hipMalloc(&mGlobalRandState, mGridSize * sizeof(hiprandState)));
    // must be managed so pattern can be loaded from file
    CUDA_ASSERT(hipMallocManaged(&grid, mGridBytes));
    CUDA_ASSERT(hipMallocManaged(&nextGrid, mGridBytes));
    CUDA_ASSERT(hipMallocManaged(&mActiveCellCount, sizeof(uint)));
    // initialization
    *mActiveCellCount = 0;
    CUDA_ASSERT(hipMemset(grid, 0, mGridBytes)); // init grid with zeros (gpu)

    // define common kernel configs
    // set_optimal_kernel_config(mGpuDeviceId);
    int tBlockSize;   // The launch configurator returned block size
    int tMinGridSize; // The minimum grid size needed to achieve the
                      // maximum occupancy for a full device launch
    int tGridSize;    // The actual grid size needed, based on input size

    /* int* minGridSize,
     * int* blockSize,
     * T func,
     * size_t dynamicSMemSize = 0,
     * int blockSizeLimit = 0 */
    hipOccupancyMaxPotentialBlockSize(&tMinGridSize, &tBlockSize,
                                       k_evolve_count_rule, 0, 0);
    // Round up according to array size
    tGridSize = (mGridSize + tBlockSize - 1) / tBlockSize;

    k_evolve_count_rule<<<tGridSize, tBlockSize>>>(
        grid, nextGrid, config::rows, config::cols, NULL, 0, false, NULL);

    hipDeviceSynchronize();

    // calculate theoretical occupancy
    int maxActiveBlocks;
    hipOccupancyMaxActiveBlocksPerMultiprocessor(
        &maxActiveBlocks, k_evolve_count_rule, tBlockSize, 0);

    float occupancy =
        (maxActiveBlocks * tBlockSize / gpuProps.warpSize) /
        (float)(gpuProps.maxThreadsPerMultiProcessor / gpuProps.warpSize);

    spdlog::info(
        "Kernel launch config set to {} blocks of size {} with occupancy of {}",
        tGridSize, tBlockSize, occupancy);

    // blocks should be a multiple of #SMs on the grid (assume #SMs is even) !
    // actually the number of blocks should
    mGpuBlocks = tGridSize; // 1156 on a 3080 - 17 blocks per SM
    // threads should be a multiple of warpSize on the block
    mGpuThreadsPerBlock = tBlockSize;

    ///////////////////////////////////////////

    // initialize RNG
    k_setup_rng<<<mGpuBlocks, mGpuThreadsPerBlock>>>(
        config::rows, config::cols, mGlobalRandState, randSeed);

    // prefetch grid to CPU so it can fill it properly
    CUDA_ASSERT(hipMemPrefetchAsync(grid, mGridBytes, hipCpuDeviceId));
    CUDA_ASSERT(hipMemPrefetchAsync(nextGrid, mGridBytes, hipCpuDeviceId));

    // create grid evolving CUDA stream
    CUDA_ASSERT(hipStreamCreate(&mEvolveStream));

    // if rendering is enabled
    if (gridVBO) {
        // create buffer updating CUDA stream
        CUDA_ASSERT(hipStreamCreate(&mBufferUpdateStream));
        // register OpenGL VBO to use with CUDA
        CUDA_ASSERT(hipGraphicsGLRegisterBuffer(
            &mGridVBOResource, *gridVBO, cudaGraphicsMapFlagsWriteDiscard));
    }

    // define the live log buffer
    mLiveLogBuffer = pLiveLogBuffer;

    spdlog::info("Automata GPU engine is ready.");
}

AutomataBase::~AutomataBase() {
    // I believe member streams are destroyed automatically (since I get
    // segfault if do it here).
    // Also, graphics resources are automatically unbound.

    // free the grid
    CUDA_ASSERT(hipFree(grid));
    CUDA_ASSERT(hipFree(nextGrid));
    CUDA_ASSERT(hipFree(mActiveCellCount));
    CUDA_ASSERT(hipFree(mGlobalRandState));
    hipDeviceReset();
}

void AutomataBase::prepare() {
    // prefetch grid data to GPU
    CUDA_ASSERT(hipMemPrefetchAsync(grid, mGridBytes, mGpuDeviceId));
    CUDA_ASSERT(hipMemPrefetchAsync(nextGrid, mGridBytes, mGpuDeviceId));
    CUDA_ASSERT(
        hipMemPrefetchAsync(mActiveCellCount, sizeof(uint), mGpuDeviceId));

    // initialize grid with fillProb
    if (config::fillProb > 0)
        k_init_grid<<<mGpuBlocks, mGpuThreadsPerBlock>>>(
            grid, config::rows, config::cols, mGlobalRandState,
            config::fillProb);
    CUDA_ASSERT(hipGetLastError());
    CUDA_ASSERT(hipDeviceSynchronize());
}

void AutomataBase::evolve(const bool logEnabled) {
    const std::chrono::steady_clock::time_point timeStart =
        std::chrono::steady_clock::now();

    run_evolution_kernel(logEnabled); // count alive cells if log is enabled
    // should I call hipDeviceSynchronize?
    CUDA_ASSERT(hipDeviceSynchronize());

    // simply swap buffers to avoid reallocation
    GridType *tmpGrid = grid;
    grid = nextGrid;
    nextGrid = tmpGrid;

    // calculate timing
    const unsigned long duration =
        std::chrono::duration_cast<std::chrono::nanoseconds>(
            std::chrono::steady_clock::now() - timeStart)
            .count();
    stats::totalEvolveTime += duration;
    // update live buffer
    if (logEnabled) {
        // bring back variable to CPU
        CUDA_ASSERT(hipMemPrefetchAsync(mActiveCellCount, sizeof(uint),
                                         hipCpuDeviceId));

        *mLiveLogBuffer << " | Evolve Kernel: " << duration
                        << " ns | Active cells: " << *mActiveCellCount;

        // reset it send it back to GPU
        *mActiveCellCount = 0;
        CUDA_ASSERT(
            hipMemPrefetchAsync(mActiveCellCount, sizeof(uint), mGpuDeviceId));
    }
}

void AutomataBase::run_evolution_kernel(const bool countAliveCells) {
    k_evolve_count_rule<<<mGpuBlocks, mGpuThreadsPerBlock, 0, mEvolveStream>>>(
        grid, nextGrid, config::rows, config::cols, mGlobalRandState,
        config::virtualFillProb, countAliveCells, mActiveCellCount);
    CUDA_ASSERT(hipGetLastError());
}

void AutomataBase::update_grid_buffers() {
#ifndef HEADLESS_ONLY

    if (!mGridVBOResource) {
        fprintf(stderr, "ERROR: Cannot call update_grid_buffers with rendering "
                        "disabled.\n");
        exit(EXIT_FAILURE);
    }

    const std::chrono::steady_clock::time_point timeStart =
        std::chrono::steady_clock::now();

    // update projection limits
    proj::update();

    // map OpenGL buffer object for writing from CUDA
    fvec2s *gridVertices;
    CUDA_ASSERT(hipGraphicsMapResources(1, &mGridVBOResource, 0));
    size_t numBytes;
    CUDA_ASSERT(hipGraphicsResourceGetMappedPointer(
        reinterpret_cast<void **>(&gridVertices), &numBytes, mGridVBOResource));
    // printf("CUDA mapped VBO: May access %ld bytes\n", numBytes);

    // launch kernels
    // reset buffers
    k_reset_grid_buffers<<<mGpuBlocks, mGpuThreadsPerBlock, 0,
                           mBufferUpdateStream>>>(
        gridVertices, proj::info.numVertices.x, proj::info.numVertices.y);
    CUDA_ASSERT(hipGetLastError());
    // update buffers
    k_update_grid_buffers<<<mGpuBlocks, mGpuThreadsPerBlock, 0,
                            mBufferUpdateStream>>>(
        grid, gridVertices, config::cols, proj::info.numVertices.x,
        proj::cellDensity, proj::gridLimX, proj::gridLimY);
    CUDA_ASSERT(hipGetLastError());

    // unmap buffer object
    CUDA_ASSERT(hipGraphicsUnmapResources(1, &mGridVBOResource, 0));

    // calculate timing
    stats::totalBufferTime +=
        std::chrono::duration_cast<std::chrono::nanoseconds>(
            std::chrono::steady_clock::now() - timeStart)
            .count();

#endif // HEADLESS_ONLY
}

} // namespace gpu